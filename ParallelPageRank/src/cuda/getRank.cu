#include <stdio.h>
#include "getRank.h"
#include "hipblas.h"
#include "hipblas.h"
#include "hipsparse.h"
#include "hip/hip_runtime.h"

/*
int main(int argc, char *argv[]){
   MKL_INT nnz = 14;
   double tol = .0001;
   double Avals[14];
   ones(Avals, nnz);
   MKL_INT rowind[14] = {0, 1, 1, 1, 2, 2, 2, 3, 3, 3, 4, 4, 4, 4};
   MKL_INT colind[14] = {1, 2, 3, 4, 1, 3, 4, 1, 2, 4, 1, 2, 3, 5};
   MKL_INT numRow = 6;
   makeP(Avals, rowind, &numRow, colind, &nnz, .95);
   //float *sinkNodes = (float*)malloc(sizeof(float)*numRow*numSinks);
   //ones(sinkNodes, numRow*numSinks);
   //MKL_INT *sinkRow = (MKL_INT*)malloc(sizeof(MKL_INT)*numRow*numSinks);
   //MKL_INT *sinkCol = (MKL_INT*)malloc(sizeof(MKL_INT)*numRow*numSinks);
   //makeSinks(sinkRow, sinkCol, d, numRow);

   double *x = (float*)malloc(sizeof(float)*numRow);
   int i;
   for(i = 0; i<numRow; i++){
      x[i] = (float)1/numRow;
   }
   getRank(Avals, x, rowind, colind, &numRow, &nnz, tol, .95);
   printf("result: \n");
   for(i = 0; i<numRow; i++){
      printf("x[%d] = %lf\n", i+1, x[i]);
   }

   free(x);
   return 0;
}
*/
void makeP(double *Avals, int *rowind, int numRow, int *colind, int nnz, double dP){
   FILE *f0 = fopen("csrAvals.txt", "w");
   FILE *f2 = fopen("csrCols.txt", "w");
   FILE *f3 = fopen("csrRow.txt", "w");
   FILE *f4 = fopen("newCsrRow.txt", "w");
   int *testRowind = (int*)calloc(sizeof(int), nnz);
   int s;
   for (s = 0; s < nnz; s++) {
      fprintf(f0,"%lf\n", Avals[s]);
      fprintf(f3, "%d\n", colind[s]);
   }
   printf("Entering MAKEP.\n");
   hipsparseStatus_t status;
   hipsparseHandle_t handle=0;
   hipsparseMatDescr_t descr=0;
   hipsparseOperation_t transa = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    
    status = hipsparseCreate(&handle);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Cusparse Library Initialization.");
      exit(2);
   }
   status = hipsparseCreateMatDescr(&descr);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Matrix descriptor initialization failed");
      exit(2);
   }
   status = hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatType failed");
      exit(2);
   }
   status = hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ONE);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatIndexBase failed");
      exit(2);
   }
   hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ONE;
   double *one = (double*)malloc(sizeof(double)*(numRow));
   double *d = (double*)calloc(numRow, sizeof(double));
   double *dev_one, *dev_d, *dev_Avals;
   int *dev_csrRowInd, *dev_colind, *dev_rowind;
   int i; 
   //int sinkNodes = 0;

   ones(one, numRow);
   //Convert rowInd vector to CSR format
   hipMalloc(&dev_rowind, sizeof(int)*(numRow));
   hipMalloc(&dev_csrRowInd, sizeof(int)*numRow+1);
   hipMemcpy(dev_rowind, rowind, sizeof(int) * (numRow), hipMemcpyHostToDevice);
   printf("Before coo2csr.\n");
   status = hipsparseXcoo2csr(handle, dev_rowind, nnz, numRow, dev_csrRowInd, idxBase);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("FAILURE to set csr row indices.");
      exit(2);
   }
   printf("after coo2csr.\n");
   hipMemcpy(testRowind, dev_rowind, sizeof(int)*(numRow), hipMemcpyDeviceToHost);
   for(i = 0; i < numRow; i++) {
      fprintf(f2,"%d\n", testRowind[s]);
   }

   hipMemcpy(rowind, dev_csrRowInd, sizeof(int)*(numRow), hipMemcpyDeviceToHost);
   for(i = 0; i < numRow; i++) {
      fprintf(f4,"%d\n", rowind[s]);
   }


   // csr format only way suportted in CUDA
   hipMalloc(&dev_one, sizeof(double)*(numRow));
   hipMalloc(&dev_d, sizeof(double)*(numRow));
   hipMalloc(&dev_Avals, sizeof(double)*(numRow));
   hipMalloc(&dev_colind, sizeof(int)*(numRow));

   
   hipMemcpy(dev_one, one, sizeof(double) * (numRow), hipMemcpyHostToDevice);
   hipMemcpy(dev_d, d, sizeof(double) * (numRow), hipMemcpyHostToDevice);
   hipMemcpy(dev_Avals, Avals, sizeof(double) * (numRow), hipMemcpyHostToDevice);
   hipMemcpy(dev_colind, colind, sizeof(int) * (numRow),  hipMemcpyHostToDevice);
   //csr multiplication call
   double alpha = 1, beta = 0;

   printf("Before csrmv.\n");
   hipsparseDcsrmv(handle, transa, numRow, numRow, nnz, &alpha, descr, 
                  dev_Avals, dev_csrRowInd, dev_colind, one, &beta, dev_d);
   printf("After csrmv.\n");
   printf("Before uncompressing row indices.\n");
   hipsparseXcsr2coo(handle, dev_csrRowInd, nnz, numRow, dev_rowind, idxBase);
   printf("After uncompressing row indices.\n");

   hipMemcpy(rowind, dev_rowind, sizeof(int) * (numRow), hipMemcpyDeviceToHost);
   hipMemcpy(colind, dev_colind, sizeof(int) * (numRow), hipMemcpyDeviceToHost);
   hipMemcpy(d, dev_d, sizeof(double) * (numRow), hipMemcpyDeviceToHost);

//   mkl_cspblas_dcoogemv (&transa, numRow, Avals ,rowind , colind , nnz , one, d );
   /*
   for(i = 0; i<*numRow; i++){
    printf("d[%d] = %lf\n", i, d[i]);
   }
   */
   FILE *f1 = fopen("csrAvalsAfter.txt", "w");

   for (i = 0; i< nnz; i++){
         if (d[rowind[i]] && Avals[i]) {
            Avals[i] = dP/d[rowind[i]];
         }
         fprintf(f1, "P[%d, %d] = %lf\n", rowind[i]+1, colind[i]+1, Avals[i]);
   }
   hipFree(dev_rowind);
   hipFree(dev_colind);
   hipFree(dev_Avals);
   hipFree(dev_one);
   hipFree(dev_d);
   free(d);
   free(one);
   printf("Leaving MAKEP.\n");
}

/*
 makeP will need to spit out d for for this to work
void makeSinks(MKL_INT *rowind, MKL_INT *colind, float *d, MKL_INT numRow){
   int i, j, count= 0;
   for (i = 0; i<numRow; i++) {
      if (!d[i]) {
         for (j = 0; j<numRow; j++) {
            rowind[count] = i;
            colind[count] = j;
            count++;
         }
      }
   }
}
*/
void getRank(double *Pvals, double *x, int *rowind, int *colind, int numRows, int nnz, double tol, double dP){
   printf("Entering getRank.\n");
      //hipsparseDcsrmv(hipsparseHandle_t handle, hipsparseOperation_t transA, int m, int n, int nnz, const double *alpha, const hipsparseMatDescr_t descrA, const double *csrValA, const int *csrRowPtrA, const int *csrColIndA, const double *x, const double *beta, double *y)
   hipsparseStatus_t status;
   hipsparseHandle_t handle=0;
   hipsparseMatDescr_t descr=0;
//   hipsparseOperation_t transa = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    
    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
            perror("CUSPARSE Library initialization failed");
           exit(2);
    }
    status = hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
           perror("Matrix descriptor initialization failed");
           exit(2);
    }
    status = hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
           perror("hipsparseSetMatType failed");
           exit(2);
    }
    status = hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
           perror("hipsparseSetMatIndexBase failed");
           exit(2);
    }
   double *dev_y;
   double *dev_x;
   double *dev_Pvals;
   int *dev_rowind;
   int *dev_colind;

   hipMalloc(&dev_y, (sizeof(double)*numRows));
   hipMalloc(&dev_x, sizeof(double)*(numRows));
   hipMalloc(&dev_Pvals, sizeof(double)*(numRows));
   hipMalloc(&dev_rowind, sizeof(double)*(numRows));
   hipMalloc(&dev_colind, sizeof(double)*(numRows));
   int i;
   double *y = (double*)malloc(sizeof(double)*numRows);
   ones(y, numRows);
   double alpha = 1, beta = 1;
   double *dev_alpha, *dev_beta;
   hipMalloc(&dev_alpha, sizeof(double)*1);
   hipMalloc(&dev_beta, sizeof(double)*1);
   hipMemcpy(dev_alpha, &alpha, sizeof(double)*1, hipMemcpyHostToDevice);
   hipMemcpy(dev_beta, &beta, sizeof(double)*1, hipMemcpyHostToDevice);

   //double error = 10.0;
   //while (error>tol) {
   printf("Before While loop.\n");
   FILE *file;
   file = fopen("cudaCSR.txt", "w");
   for (i = 0; i < nnz; i++) {
      fprintf(file, "Pvals[%d] = %lf.\n", i, Pvals[i]);
   } 
   i = 0;
   while(i++<500){
      printf("i = %d.\n", i);
      beta = (double)((1-dP)/(numRows));
      //launch kernel
      //hipsparseDcsrmv(handle, transa, numRows, numRows, nnz, dev_alpha, descr, dev_Pvals, dev_rowind, dev_colind, dev_x, dev_beta, dev_y);
      printf("After cusparseDCsrmv.\n");
      /*hipMemcpy(x, dev_x, sizeof(double)*(numRows));
      hipMemcpy(y, dev_y, sizeof(double)*(numRows));
      error = getError(x, y, numRows);*/
      hipMemcpy(dev_x, dev_y, numRows*sizeof(double), hipMemcpyDeviceToDevice);
      ones(y, numRows);
      //printf("error: %lf\n", error);
   }
   printf("After while loop.\n");
   hipMemcpy(x, dev_x, sizeof(double)*numRows, hipMemcpyDeviceToHost);
   hipFree(dev_x);
   hipFree(dev_y);
   hipFree(dev_Pvals);
   hipFree(dev_rowind);
   hipFree(dev_colind);
   free(y);
   printf("Leaving getRank.\n");
}

double sum(double *x, int N){
   printf("Entering sum.\n");
   int i;
   double result = 0;
//#pragma omp parallel for simd reduction(+:result)
   for (i = 0; i<N; i++){
      result+= x[i];
   }
   printf("Leaving sum.\n");
   return result;
}

void ones(double *a, int N){
   printf("Entering ones.\n");
   int i;
//#pragma omp parallel for simd
   for (i =0; i< N; i++) {
      a[i] = 1;
   }
   printf("Leaving ones.\n");
}

double getError(double *v1, double *v2, int size){
   
   int i;
   double result;
 //  #pragma omp parallel for simd
   for (i = 0; i<size; i++) {
      v1[i] = v1[i]-v2[i];
   }
   result = 10; // not using this function to terminate while loop currently.
   return result;
}
