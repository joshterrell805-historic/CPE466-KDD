#include <stdio.h>
#include "getRank.h"
#include "/usr/include/cuda/hip/hip_runtime.h"
#include <hipblas.h>
#include <hipblas.h>
#include <hipsparse.h>

void makeP(double *Avals, int *rowind, int numRow, int *colind, int nnz, double dP){
   printf("Entering MAKEP.\n");
   hipsparseStatus_t status;
   hipsparseHandle_t handle=0;
   hipsparseMatDescr_t descr=0;
   hipsparseOperation_t transa = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    
   status = hipsparseCreate(&handle);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Cusparse Library Initialization.");
      exit(2);
   }
   status = hipsparseCreateMatDescr(&descr);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Matrix descriptor initialization failed");
      exit(2);
   }
   status = hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatType failed");
      exit(2);
   }
   status = hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatIndexBase failed");
      exit(2);
   }
   hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
   double *one = (double*)malloc(sizeof(double)*(numRow));
   double *d = (double*)malloc(numRow*sizeof(double));
   double *dev_one, *dev_d, *dev_Avals;
   int *dev_csrRowInd, *dev_colind, *dev_rowind;
 //  int *csrRowInd;
   int i;

//   csrRowInd = (int*)malloc(sizeof(int)*(numRow+1));

   //Convert rowInd vector to CSR format
   hipMalloc(&dev_rowind, sizeof(int)*(nnz));
   hipMalloc(&dev_csrRowInd, sizeof(int)*(numRow+1));
   hipMemcpy(dev_rowind, rowind, sizeof(int) * (nnz), hipMemcpyHostToDevice);

   printf("Before coo2csr.\n");
   status = hipsparseXcoo2csr(handle, dev_rowind, nnz, numRow, dev_csrRowInd, idxBase);
   
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("FAILURE to set csr row indices.");
      exit(2);
   }
   printf("after coo2csr.\n");
//   FILE *fileCSR = fopen("newCSRarray.txt", "w");
//   hipMemcpy(csrRowInd, dev_csrRowInd, sizeof(int)*(numRow+1), hipMemcpyDeviceToHost);
//   for(i = 0; i < numRow + 1; i++) {
//      fprintf(fileCSR,"%d\n", csrRowInd[i]);
//   }


   ones(one, numRow);
   ones(d, numRow);
   // csr format only way suportted in CUDA
   hipMalloc(&dev_one, sizeof(double)*(numRow));
   hipMalloc(&dev_d, sizeof(double)*(numRow));
   hipMalloc(&dev_Avals, sizeof(double)*(nnz));
   hipMalloc(&dev_colind, sizeof(int)*(nnz));

   hipMemcpy(dev_d, d, sizeof(double) * (numRow), hipMemcpyHostToDevice);
   hipMemcpy(dev_one, one, sizeof(double) * (numRow), hipMemcpyHostToDevice);
   hipMemcpy(dev_Avals, Avals, sizeof(double) * (nnz), hipMemcpyHostToDevice);
   hipMemcpy(dev_colind, colind, sizeof(int) * (nnz),  hipMemcpyHostToDevice);
   //csr multiplication call
   double alpha = 1, beta = 0;

   printf("Before csrmv.\n");
   hipsparseDcsrmv(handle, transa, numRow, numRow, nnz, &alpha, descr, 
                  dev_Avals, dev_csrRowInd, dev_colind, dev_one, &beta, dev_d);

   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("FAILURE to makeP.");
      exit(2);
   }
   printf("After csrmv.\n");
   
   hipMemcpy(Avals, dev_Avals, sizeof(int) * (nnz), hipMemcpyDeviceToHost);
   hipMemcpy(one, dev_one, sizeof(double) * (numRow), hipMemcpyDeviceToHost);
   hipMemcpy(d, dev_d, sizeof(double) * (numRow), hipMemcpyDeviceToHost);

   for (i = 0; i< nnz; i++){
         if (d[rowind[i]] && Avals[i]) {
            Avals[i] = dP/d[rowind[i]];
         }
//         fprintf(f1, "P[%d, %d] = %lf\n", rowind[i]+1, colind[i]+1, Avals[i]);
   }
   hipFree(dev_rowind);
   hipFree(dev_colind);
   hipFree(dev_Avals);
   hipFree(dev_one);
   hipFree(dev_d);
   free(d);
   free(one);
   printf("Leaving MAKEP.\n");
}

void getRank(double *Pvals, double *x, int *rowind, int *colind, int numRows, int nnz, double tol, double dP){
   printf("Entering getRank.\n");
   hipsparseStatus_t status;
   hipsparseHandle_t handle=0;
   hipsparseMatDescr_t descr=0;
   hipsparseOperation_t transa = HIPSPARSE_OPERATION_TRANSPOSE;
    
   status = hipsparseCreate(&handle);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Failed to create handle.");
      exit(2);
   }
   status = hipsparseCreateMatDescr(&descr);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Matrix descriptor initialization failed");
      exit(2);
   }
   status = hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatType failed");
      exit(2);
   }
   status = hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatIndexBase failed");
      exit(2);
   }
   hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
   double *dev_y;
   double *dev_x;
   double *dev_Pvals;
   int *dev_rowind, *dev_csrRowInd, *dev_colind;
   int i;
   double *y = (double*)malloc(sizeof(double)*numRows);
   double alpha = 1, beta;
   //double error = 10.0;

   ones(y, numRows);

   hipMalloc(&dev_y, (sizeof(double)*numRows));
   hipMalloc(&dev_x, sizeof(double)*(numRows));
   hipMalloc(&dev_Pvals, sizeof(double)*(nnz));
   hipMalloc(&dev_rowind, sizeof(double)*(nnz));
   hipMalloc(&dev_csrRowInd, sizeof(double)*(numRows+1));
   hipMalloc(&dev_colind, sizeof(double)*(nnz));

   hipMemcpy(dev_y, y, sizeof(double)*(numRows), hipMemcpyHostToDevice);
   hipMemcpy(dev_x, x, sizeof(double)*(numRows), hipMemcpyHostToDevice);
   hipMemcpy(dev_rowind, rowind, sizeof(double)*(nnz), hipMemcpyHostToDevice);
   hipMemcpy(dev_colind, colind, sizeof(double)*(nnz), hipMemcpyHostToDevice);
   hipMemcpy(dev_Pvals, Pvals, sizeof(double)*(nnz), hipMemcpyHostToDevice);

   status = hipsparseXcoo2csr(handle, dev_rowind, nnz, numRows, dev_csrRowInd, idxBase);
   
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("FAILURE to set csr row indices.");
      exit(2);
   }

   int *csrRowInd = (int*) malloc(sizeof(int)*(numRows+1));
   hipMemcpy(csrRowInd, dev_csrRowInd, sizeof(int) *(numRows + 1), hipMemcpyDeviceToHost);
   //while (error>tol) {
   printf("Before While loop.\n");
//   FILE *file;
//   file = fopen("cudaCSR.txt", "w");
//   for (i = 0; i < numRows; i++) {
//      printf("csrRowInd[%d] = %d\t", i, csrRowInd[i]);
//      printf("x = %lf, y = %lf\n", x[i], y[i]);
//   }
//   for (i = 0; i < nnz; i++) {
//      fprintf(file, "Pvals[%d] = %lf.\n", i, Pvals[i]);
//   } 
   i = 0;
//   int j;
   while(i++<20){
      printf("i = %d.\n", i);
      beta = (double)((1-dP)/(numRows));
      printf("Before hipsparseDcsrmv.\n");
      hipsparseDcsrmv(handle, transa, numRows, numRows, nnz, &alpha, descr, dev_Pvals,
                     dev_csrRowInd, dev_colind, dev_x, &beta, dev_y);
      printf("After cusparseDCsrmv.\n\n\n\n");
      hipMemcpy(x, dev_x, sizeof(double)*(numRows), hipMemcpyDeviceToHost);
      hipMemcpy(y, dev_y, sizeof(double)*(numRows), hipMemcpyDeviceToHost);
//   for (j = 0; j < numRows; j++) {
//      printf("csrRowInd[%d] = %d\t", j, csrRowInd[j]);
//      printf("x = %lf, y = %lf\n", x[j], y[j]);
//   }
      //error = getError(x, y, numRows);
      hipMemcpy(dev_x, dev_y, numRows*sizeof(double), hipMemcpyDeviceToDevice);
      ones(y, numRows);
      //printf("error: %lf\n", error);
   }
   printf("After while loop.\n");
   hipMemcpy(x, dev_x, sizeof(double)*numRows, hipMemcpyDeviceToHost);
   hipFree(dev_x);
   hipFree(dev_y);
   hipFree(dev_Pvals);
   hipFree(dev_rowind);
   hipFree(dev_colind);
   free(y);
   printf("Leaving getRank.\n");
}

double sum(double *x, int N){
   printf("Entering sum.\n");
   int i;
   double result = 0;
//#pragma omp parallel for simd reduction(+:result)
   for (i = 0; i<N; i++){
      result+= x[i];
   }
   printf("Leaving sum.\n");
   return result;
}

void ones(double *a, int N){
   printf("Entering ones.\n");
   int i;
//#pragma omp parallel for simd
   for (i =0; i< N; i++) {
      a[i] = 1;
   }
   printf("Leaving ones.\n");
}

double getError(double *v1, double *v2, int size){
   
   int i;
   double result;
 //  #pragma omp parallel for simd
   for (i = 0; i<size; i++) {
      v1[i] = v1[i]-v2[i];
   }
   result = 10; // not using this function to terminate while loop currently.
   return result;
}
