#include <stdio.h>
#include "getRank.h"
#include "/usr/include/cuda/hip/hip_runtime.h"
#include <hipblas.h>
#include <hipsparse.h>

void makeP(double *Avals, int *rowind, int numRow, int *colind, int nnz, double dP){
   hipsparseStatus_t status;
   hipsparseHandle_t handle=0;
   hipsparseMatDescr_t descr=0;
   hipsparseOperation_t transa = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    
   status = hipsparseCreate(&handle);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Cusparse Library Initialization.");
      exit(2);
   }
   status = hipsparseCreateMatDescr(&descr);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Matrix descriptor initialization failed");
      exit(2);
   }
   status = hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatType failed");
      exit(2);
   }
   status = hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatIndexBase failed");
      exit(2);
   }
   hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
   double *one = (double*)malloc(sizeof(double)*(numRow));
   double *d = (double*)malloc(numRow*sizeof(double));
   double *dev_one, *dev_d, *dev_Avals;
   int *dev_csrRowInd, *dev_colind, *dev_rowind;
   int i;

   dP = .95;
   //Convert rowInd vector to CSR format
   hipMalloc(&dev_rowind, sizeof(int)*(nnz));
   hipMalloc(&dev_csrRowInd, sizeof(int)*(numRow+1));
   hipMemcpy(dev_rowind, rowind, sizeof(int) * (nnz), hipMemcpyHostToDevice);

   status = hipsparseXcoo2csr(handle, dev_rowind, nnz, numRow, dev_csrRowInd, idxBase);
   
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("FAILURE to set csr row indices.");
      exit(2);
   }


   ones(one, numRow);
   ones(d, numRow);
   // csr format only way suportted in CUDA
   hipMalloc(&dev_one, sizeof(double)*(numRow));
   hipMalloc(&dev_d, sizeof(double)*(numRow));
   hipMalloc(&dev_Avals, sizeof(double)*(nnz));
   hipMalloc(&dev_colind, sizeof(int)*(nnz));

   hipMemcpy(dev_d, d, sizeof(double) * (numRow), hipMemcpyHostToDevice);
   hipMemcpy(dev_one, one, sizeof(double) * (numRow), hipMemcpyHostToDevice);
   hipMemcpy(dev_Avals, Avals, sizeof(double) * (nnz), hipMemcpyHostToDevice);
   hipMemcpy(dev_colind, colind, sizeof(int) * (nnz),  hipMemcpyHostToDevice);
   //csr multiplication call
   double alpha = 1, beta = 0;

   hipsparseDcsrmv(handle, transa, numRow, numRow, nnz, &alpha, descr, 
                  dev_Avals, dev_csrRowInd, dev_colind, dev_one, &beta, dev_d);

   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("FAILURE to makeP.");
      exit(2);
   }
   
   hipMemcpy(Avals, dev_Avals, sizeof(int) * (nnz), hipMemcpyDeviceToHost);
   hipMemcpy(one, dev_one, sizeof(double) * (numRow), hipMemcpyDeviceToHost);
   hipMemcpy(d, dev_d, sizeof(double) * (numRow), hipMemcpyDeviceToHost);

   for (i = 0; i< nnz; i++){
         if (d[rowind[i]] && Avals[i]) {
            Avals[i] = dP/d[rowind[i]];
         }
   }
   hipFree(dev_rowind);
   hipFree(dev_colind);
   hipFree(dev_Avals);
   hipFree(dev_one);
   hipFree(dev_d);
   free(d);
   free(one);
}

void getRank(double *Pvals, double *x, int *rowind, int *colind, int numRows, int nnz, double tol, double dP){
   hipsparseStatus_t status;
   hipsparseHandle_t handle=0;
   hipsparseMatDescr_t descr=0;
   hipsparseOperation_t transa = HIPSPARSE_OPERATION_TRANSPOSE;
    
   status = hipsparseCreate(&handle);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Failed to create handle.");
      exit(2);
   }
   status = hipsparseCreateMatDescr(&descr);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Matrix descriptor initialization failed");
      exit(2);
   }
   status = hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatType failed");
      exit(2);
   }
   status = hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatIndexBase failed");
      exit(2);
   }
   hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
   double *dev_y;
   double *dev_x;
   double *dev_Pvals;
   int *dev_rowind, *dev_csrRowInd, *dev_colind;

   int i;
   double *y = (double*)malloc(sizeof(double)*numRows);
   double *alpha, *beta;
   alpha = (double*) malloc(sizeof(double));
   beta = (double*) malloc(sizeof(double));
   alpha[0] = 1;

   //double error = 10.0;

   ones(y, numRows);

   hipMalloc(&dev_y, (sizeof(double)*numRows));
   hipMalloc(&dev_x, sizeof(double)*(numRows));
   hipMalloc(&dev_Pvals, sizeof(double)*(nnz));
   hipMalloc(&dev_rowind, sizeof(double)*(nnz));
   hipMalloc(&dev_csrRowInd, sizeof(double)*(numRows+1));
   hipMalloc(&dev_colind, sizeof(double)*(nnz));

   hipMemcpy(dev_y, y, sizeof(double)*(numRows), hipMemcpyHostToDevice);
   hipMemcpy(dev_x, x, sizeof(double)*(numRows), hipMemcpyHostToDevice);
   hipMemcpy(dev_rowind, rowind, sizeof(double)*(nnz), hipMemcpyHostToDevice);
   hipMemcpy(dev_colind, colind, sizeof(double)*(nnz), hipMemcpyHostToDevice);
   hipMemcpy(dev_Pvals, Pvals, sizeof(double)*(nnz), hipMemcpyHostToDevice);
   

   status = hipsparseXcoo2csr(handle, dev_rowind, nnz, numRows, dev_csrRowInd, idxBase);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("FAILURE to set csr row indices.");
      exit(2);
   }

   
   i = 0;
//   while (error>tol) {
   while(i++<50){
      //i++;
      beta[0] = (double)((1-dP)/(numRows));
      hipsparseDcsrmv(handle, transa, numRows, numRows, nnz, alpha, descr, dev_Pvals,
                     dev_csrRowInd, dev_colind, dev_x, beta, dev_y);
     // error = hipblasDnrm2(numRows, dev_y, 1);
      hipMemcpy(dev_x, dev_y, numRows*sizeof(double), hipMemcpyDeviceToDevice);
      hipMemcpy(dev_y, y, sizeof(double) * numRows, hipMemcpyHostToDevice);
   }
   hipMemcpy(x, dev_x, sizeof(double)*numRows, hipMemcpyDeviceToHost);
   hipFree(dev_x);
   hipFree(dev_y);
   hipFree(dev_Pvals);
   hipFree(dev_rowind);
   hipFree(dev_colind);
   free(y);
}

double sum(double *x, int N){
   int i;
   double result = 0;
//#pragma omp parallel for simd reduction(+:result)
   for (i = 0; i<N; i++){
      result+= x[i];
   }
   return result;
}

void ones(double *a, int N){
   int i;
//#pragma omp parallel for simd
   for (i =0; i< N; i++) {
      a[i] = 1;
   }
}

double getError(double *v1, double *v2, int size){
   
   int i;
   double result;
 //  #pragma omp parallel for simd
   for (i = 0; i<size; i++) {
      v1[i] = v1[i]-v2[i];
   }
   result = 10; // not using this function to terminate while loop currently.
   double *dev_v1;
   hipMalloc(&dev_v1, sizeof(double)*size);

   return result;
}
