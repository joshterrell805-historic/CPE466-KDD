#include <stdio.h>
#include "getRank.h"
#include "/usr/include/cuda/hip/hip_runtime.h"
#include <hipblas.h>
#include <hipblas.h>
#include <hipsparse.h>

void makeP(double *Avals, int *rowind, int numRow, int *colind, int nnz, double dP){
   FILE *f0 = fopen("csrAvals.txt", "w");
   FILE *f2 = fopen("csrCols.txt", "w");
   FILE *f3 = fopen("csrRow.txt", "w");
   FILE *f4 = fopen("newCsrRow.txt", "w");
   FILE *f5 = fopen("newCsrRow2.txt", "w");
   printf("dp = %lf\n", dP);
   printf("Entering MAKEP.\n");
   int i;
   for (i = 0; i < nnz; i++) {
      fprintf(f3, "i = %d, %d\n",i, rowind[i]);
      fprintf(f2, "i = %d, %d\n",i, colind[i]);
      fprintf(f0, "i = %d, %lf\n",i, Avals[i]);
   }

   hipsparseStatus_t status;
   hipsparseHandle_t handle=0;
   hipsparseMatDescr_t descr=0;
   hipsparseOperation_t transa = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    
    status = hipsparseCreate(&handle);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Cusparse Library Initialization.");
      exit(2);
   }
   status = hipsparseCreateMatDescr(&descr);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("Matrix descriptor initialization failed");
      exit(2);
   }
   status = hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatType failed");
      exit(2);
   }
   status = hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("hipsparseSetMatIndexBase failed");
      exit(2);
   }
   hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
   double *one = (double*)malloc(sizeof(double)*(numRow));
   double *d = (double*)malloc(numRow*sizeof(double));
   double *dev_one, *dev_d, *dev_Avals;
   int *dev_csrRowInd, *dev_colind, *dev_rowind, *csrRowInd;
   csrRowInd = (int*)malloc(sizeof(int)*(numRow+1));

   //Convert rowInd vector to CSR format
   hipMalloc(&dev_rowind, sizeof(int)*(nnz));
   hipMalloc(&dev_csrRowInd, sizeof(int)*(numRow+1));
   hipMemcpy(dev_rowind, rowind, sizeof(int) * (nnz), hipMemcpyHostToDevice);
   printf("Before coo2csr.\n");
   status = hipsparseXcoo2csr(handle, dev_rowind, nnz, numRow, dev_csrRowInd, idxBase);
   printf("status of hipsparseXcoo2csr is: %d.\n",status);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("FAILURE to set csr row indices.");
      exit(2);
   }
   printf("after coo2csr.\n");
   hipMemcpy(csrRowInd, dev_csrRowInd, sizeof(int)*(numRow+1), hipMemcpyDeviceToHost);
   for(i = 0; i < numRow + 1; i++) {
      fprintf(f4,"%d\n", csrRowInd[i]);
   }


   ones(one, numRow);
   ones(d, numRow);
   // csr format only way suportted in CUDA
   hipMalloc(&dev_one, sizeof(double)*(numRow));
   hipMalloc(&dev_d, sizeof(double)*(numRow));
   hipMalloc(&dev_Avals, sizeof(double)*(nnz));
   hipMalloc(&dev_colind, sizeof(int)*(nnz));

   hipMemcpy(dev_d, d, sizeof(double) * (numRow), hipMemcpyHostToDevice);
   hipMemcpy(dev_one, one, sizeof(double) * (numRow), hipMemcpyHostToDevice);
   hipMemcpy(dev_Avals, Avals, sizeof(double) * (nnz), hipMemcpyHostToDevice);
   hipMemcpy(dev_colind, colind, sizeof(int) * (nnz),  hipMemcpyHostToDevice);
   //csr multiplication call
   double alpha = 1, beta = 0;

   printf("Before csrmv.\n");
   hipsparseDcsrmv(handle, transa, numRow, numRow, nnz, &alpha, descr, 
                  dev_Avals, dev_csrRowInd, dev_colind, one, &beta, dev_d);
   printf("status of hipsparseDcsrmv is: %d.\n",status);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      perror("FAILURE to makeP.");
      exit(2);
   }
   printf("After csrmv.\n");
   hipMemcpy(Avals, dev_Avals, sizeof(int) * (nnz), hipMemcpyDeviceToHost);
   hipMemcpy(one, dev_one, sizeof(double) * (numRow), hipMemcpyDeviceToHost);
   hipMemcpy(d, dev_d, sizeof(double) * (numRow), hipMemcpyDeviceToHost);
   //for (i = 0; i < nnz; i++) {
   //   printf("Avals[%d, %d] = %lf\n", rowind[i], colind[i], Avals[i]);
   //}
   for (i = 0; i < numRow; i++) {
      printf("d[%d] = %lf\n",i, d[i]);
      printf("one[%d] = %lf\n",i, one[i]);
      printf("Avals[%d] = %lf",i, Avals[i]);
   }
   //printf("Before uncompressing row indices.\n");
   //hipsparseXcsr2coo(handle, dev_csrRowInd, nnz, numRow, dev_rowind, idxBase);
   //printf("After uncompressing row indices.\n");


   FILE *f1 = fopen("csrAvalsAfter.txt", "w");

   for (i = 0; i< nnz; i++){
         if (d[rowind[i]] && Avals[i]) {
            Avals[i] = dP/d[rowind[i]];
         }
         fprintf(f1, "P[%d, %d] = %lf\n", rowind[i]+1, colind[i]+1, Avals[i]);
   }
   hipFree(dev_rowind);
   hipFree(dev_colind);
   hipFree(dev_Avals);
   hipFree(dev_one);
   hipFree(dev_d);
   free(d);
   free(one);
   printf("Leaving MAKEP.\n");
}

/*
 makeP will need to spit out d for for this to work
void makeSinks(MKL_INT *rowind, MKL_INT *colind, float *d, MKL_INT numRow){
   int i, j, count= 0;
   for (i = 0; i<numRow; i++) {
      if (!d[i]) {
         for (j = 0; j<numRow; j++) {
            rowind[count] = i;
            colind[count] = j;
            count++;
         }
      }
   }
}
*/
void getRank(double *Pvals, double *x, int *rowind, int *colind, int numRows, int nnz, double tol, double dP){
   printf("Entering getRank.\n");
      //hipsparseDcsrmv(hipsparseHandle_t handle, hipsparseOperation_t transA, int m, int n, int nnz, const double *alpha, const hipsparseMatDescr_t descrA, const double *csrValA, const int *csrRowPtrA, const int *csrColIndA, const double *x, const double *beta, double *y)
   hipsparseStatus_t status;
   hipsparseHandle_t handle=0;
   hipsparseMatDescr_t descr=0;
//   hipsparseOperation_t transa = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    
    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
            perror("CUSPARSE Library initialization failed");
           exit(2);
    }
    status = hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
           perror("Matrix descriptor initialization failed");
           exit(2);
    }
    status = hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
           perror("hipsparseSetMatType failed");
           exit(2);
    }
    status = hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
           perror("hipsparseSetMatIndexBase failed");
           exit(2);
    }
   double *dev_y;
   double *dev_x;
   double *dev_Pvals;
   int *dev_rowind;
   int *dev_colind;

   hipMalloc(&dev_y, (sizeof(double)*numRows));
   hipMalloc(&dev_x, sizeof(double)*(numRows));
   hipMalloc(&dev_Pvals, sizeof(double)*(numRows));
   hipMalloc(&dev_rowind, sizeof(double)*(numRows));
   hipMalloc(&dev_colind, sizeof(double)*(numRows));
   int i;
   double *y = (double*)malloc(sizeof(double)*numRows);
   ones(y, numRows);
   double alpha = 1, beta = 1;
   double *dev_alpha, *dev_beta;
   hipMalloc(&dev_alpha, sizeof(double)*1);
   hipMalloc(&dev_beta, sizeof(double)*1);
   hipMemcpy(dev_alpha, &alpha, sizeof(double)*1, hipMemcpyHostToDevice);
   hipMemcpy(dev_beta, &beta, sizeof(double)*1, hipMemcpyHostToDevice);

   //double error = 10.0;
   //while (error>tol) {
   printf("Before While loop.\n");
   FILE *file;
   file = fopen("cudaCSR.txt", "w");
   for (i = 0; i < nnz; i++) {
      fprintf(file, "Pvals[%d] = %lf.\n", i, Pvals[i]);
   } 
   i = 0;
   while(i++<500){
      printf("i = %d.\n", i);
      beta = (double)((1-dP)/(numRows));
      //launch kernel
      //hipsparseDcsrmv(handle, transa, numRows, numRows, nnz, dev_alpha, descr, dev_Pvals, dev_rowind, dev_colind, dev_x, dev_beta, dev_y);
      printf("After cusparseDCsrmv.\n");
      /*hipMemcpy(x, dev_x, sizeof(double)*(numRows));
      hipMemcpy(y, dev_y, sizeof(double)*(numRows));
      error = getError(x, y, numRows);*/
      hipMemcpy(dev_x, dev_y, numRows*sizeof(double), hipMemcpyDeviceToDevice);
      ones(y, numRows);
      //printf("error: %lf\n", error);
   }
   printf("After while loop.\n");
   hipMemcpy(x, dev_x, sizeof(double)*numRows, hipMemcpyDeviceToHost);
   hipFree(dev_x);
   hipFree(dev_y);
   hipFree(dev_Pvals);
   hipFree(dev_rowind);
   hipFree(dev_colind);
   free(y);
   printf("Leaving getRank.\n");
}

double sum(double *x, int N){
   printf("Entering sum.\n");
   int i;
   double result = 0;
//#pragma omp parallel for simd reduction(+:result)
   for (i = 0; i<N; i++){
      result+= x[i];
   }
   printf("Leaving sum.\n");
   return result;
}

void ones(double *a, int N){
   printf("Entering ones.\n");
   int i;
//#pragma omp parallel for simd
   for (i =0; i< N; i++) {
      a[i] = 1;
   }
   printf("Leaving ones.\n");
}

double getError(double *v1, double *v2, int size){
   
   int i;
   double result;
 //  #pragma omp parallel for simd
   for (i = 0; i<size; i++) {
      v1[i] = v1[i]-v2[i];
   }
   result = 10; // not using this function to terminate while loop currently.
   return result;
}
